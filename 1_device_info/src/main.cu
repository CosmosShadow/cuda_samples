#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %lu.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %lu.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %lu.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %lu.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %lu.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        std::cout << "no cuda device found." << std::endl;
        return false;
    }

    int i, index;

    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            printDeviceProp(prop);
            if (prop.major >= 1) {
                index = i;
            }
        }
    }

    hipSetDevice(index);
    std::cout << "cuda initialized with set device " << index << std::endl;
    return true;
}

int main() 
{
    if (!InitCUDA()) { 
        return 0; 
    }
    std::cout << "cuda initialized." << std::endl;
    return 0;
}







