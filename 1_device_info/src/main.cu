#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        std::cout << "no cuda device found." << std::endl;
        return false;
    }

    int i, index;

    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            std::cout << "device " << i << " version: " << prop.major << "." << prop.minor << std::endl;
            if (prop.major >= 1) {
                index = i;
            }
        }
    }

    hipSetDevice(index);
    std::cout << "cuda initialized with set device " << index << std::endl;
    return true;
}

int main() 
{
    if (!InitCUDA()) { 
        return 0; 
    }
    std::cout << "cuda initialized." << std::endl;
    return 0;
}







