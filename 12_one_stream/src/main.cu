#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include <iostream>
#include <stdio.h>  
#include <math.h>  

#define N (1024*1024)  
#define FULL_DATA_SIZE N*20  


//CUDA 初始化
bool InitCUDA()
{
    int count;

    //取得支持Cuda的装置的数目
    hipGetDeviceCount(&count);

    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;

    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

bool isSupportOverlap()
{
    //获取设备属性
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);

    //检查设备是否支持重叠功能
    if (!prop.deviceOverlap){
        printf("No device will handle overlaps. so no speed up from stream.\n");
        return false;
    } else {
        return true;
    }
}


__global__ void kernel(int* a, int *b, int*c)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID < N)
    {
        c[threadID] = (a[threadID] + b[threadID]) / 2;
    }
}

int main()
{
    if (!InitCUDA() | !isSupportOverlap()) {
        return 0;
    }

    //创建一个CUDA流
    hipStream_t stream;
    hipStreamCreate(&stream);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    //在GPU上分配内存
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    //在CPU上分配页锁定内存
    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    //主机上的内存赋值
    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = i;
        host_b[i] = FULL_DATA_SIZE - i;
    }

    //启动计时器
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);

        kernel << <N / 1024, 1024, 0, stream >> > (dev_a, dev_b, dev_c);

        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }

    // wait until gpu execution finish  
    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "消耗时间： " << elapsedTime << std::endl;

    //输出前10个结果
    for (int i = 0; i < 10; i++)
    {
        std::cout << host_c[i] << std::endl;
    }

    // free stream and mem  
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipStreamDestroy(stream);
    return 0;
}