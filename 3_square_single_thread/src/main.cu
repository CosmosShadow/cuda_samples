#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576

int data[DATA_SIZE];

//产生大量0-9之间的随机数
void GenerateNumbers(int *number, int size)
{
    for (int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

//CUDA 初始化
bool InitCUDA()
{
    int count;

    //取得支持Cuda的装置的数目
    hipGetDeviceCount(&count);

    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;

    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}


// __global__ 函数 (GPU上执行) 计算立方和
__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{
    int sum = 0;

    int i;

    clock_t start = clock();

    for (i = 0; i < DATA_SIZE; i++) {

        sum += num[i] * num[i] * num[i];

    }

    *result = sum;

    *time = clock() - start;

}


int cudaGetClockRate()
{
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, 0) == hipSuccess){
        return prop.clockRate * 1000;
    } else {
        std::cout << "cudaGetClockRate fails" << std::endl;
        return 10^9;
    }
}


int main()
{

    //CUDA 初始化
    if (!InitCUDA()) {
        return 0;
    }

    //生成随机数
    GenerateNumbers(data, DATA_SIZE);

    /*把数据复制到显卡内存中*/
    int* gpudata, *result;

    clock_t* time;

    //hipMalloc 取得一块显卡内存 ( 其中result用来存储计算结果，time用来存储运行时间 )
    hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int));
    hipMalloc((void**)&time, sizeof(clock_t));

    //hipMemcpy 将产生的随机数复制到显卡内存中
    //hipMemcpyHostToDevice - 从内存复制到显卡内存
    //hipMemcpyDeviceToHost - 从显卡内存复制到内存
    hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

    // 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
    sumOfSquares << <1, 1, 0 >> >(gpudata, result, time);


    /*把结果从显示芯片复制回主内存*/

    int sum;
    clock_t time_used;

    //hipMemcpy 将结果从显存中复制回内存
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);

    //Free
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    // 1582000 * 1000的主频
    int clockRate = cudaGetClockRate();
    printf("GPUsum: %d time: %fs\n", sum, time_used * 1.0 / clockRate);

    sum = 0;

    for (int i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i] * data[i];
    }

    printf("CPUsum: %d \n", sum);

    return 0;
}